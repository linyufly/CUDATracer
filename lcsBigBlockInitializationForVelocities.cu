/******************************************************************
File		:	lcsBigBlockInitializationForVelocities.cu
Author		:	Mingcheng Chen
Last Update	:	January 30th, 2013
*******************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 512

__global__ void BigBlockInitializationForVelocitiesKernel(double *globalStartVelocities,
			     			  double *globalEndVelocities,
			
			     			  int *blockedGlobalPointIDs,

   			     			  int *startOffsetInPoint,

			     			  int *startOffsetInPointForBig,
			     			  double *startVelocitiesForBig,
			     			  double *endVelocitiesForBig,

			     			  int *bigBlocks
			     			  ) {
	// Get work group ID
	int workGroupID = blockIdx.x;
	
	// Get number of threads in a work group
	int numOfThreads = blockDim.x;

	// Get local thread ID
	int localID = threadIdx.x;

	// Get interesting block ID of the current big block
	int interestingBlockID = bigBlocks[workGroupID];

	// Declare some work arrays
	double *gStartVelocities;
	double *gEndVelocities;
		
	int startPoint = startOffsetInPoint[interestingBlockID];

	int numOfPoints = startOffsetInPoint[interestingBlockID + 1] - startPoint;

	int startPointForBig = startOffsetInPointForBig[interestingBlockID];

	// Initialize startVelocities and endVelocities
	gStartVelocities = startVelocitiesForBig + startPointForBig * 3;
	gEndVelocities = endVelocitiesForBig + startPointForBig * 3;

	for (int i = localID; i < numOfPoints * 3; i += numOfThreads) {
		int localPointID = i / 3;
		int dimensionID = i % 3;
		int globalPointID = blockedGlobalPointIDs[startPoint + localPointID];

		gStartVelocities[i] = globalStartVelocities[globalPointID * 3 + dimensionID];
		gEndVelocities[i] = globalEndVelocities[globalPointID * 3 + dimensionID];
	}
}

extern "C"
void BigBlockInitializationForVelocities(double *globalStartVelocities,
			     			  double *globalEndVelocities,
			
			     			  int *blockedGlobalPointIDs,

   			     			  int *startOffsetInPoint,

			     			  int *startOffsetInPointForBig,
			     			  double *startVelocitiesForBig,
			     			  double *endVelocitiesForBig,

			     			  int *bigBlocks, int numOfBigBlocks
			     			  ) {
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(numOfBigBlocks, 1, 1);

	BigBlockInitializationForVelocitiesKernel<<<dimGrid, dimBlock>>>(globalStartVelocities, globalEndVelocities, blockedGlobalPointIDs,
									startOffsetInPoint, startOffsetInPointForBig, startVelocitiesForBig,
									endVelocitiesForBig, bigBlocks);

	hipError_t err = hipDeviceSynchronize();
	if (err) {
		hipGetErrorString(err);
		exit(0);
	}
}
