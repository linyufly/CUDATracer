#include "hip/hip_runtime.h"
/******************************************************************
File			:		lcsBlockedTracingKernel.cu
Author			:		Mingcheng Chen
Last Update		:		October 2nd, 2012
*******************************************************************/

#include ""
#include "CUDAKernels.h"

#include "stdio.h"

__device__ inline double DeterminantThree(double *a) {
	// a[0] a[1] a[2]
	// a[3] a[4] a[5]
	// a[6] a[7] a[8]
	return a[0] * a[4] * a[8] + a[1] * a[5] * a[6] + a[2] * a[3] * a[7] -
		   a[0] * a[5] * a[7] - a[1] * a[3] * a[8] - a[2] * a[4] * a[6];
}

__device__ inline void CalculateNaturalCoordinates(double X, double Y, double Z,
										double *tetX, double *tetY, double *tetZ, double *coordinates) {
	X -= tetX[0];
	Y -= tetY[0];
	Z -= tetZ[0];

	double det[9] = {tetX[1] - tetX[0], tetY[1] - tetY[0], tetZ[1] - tetZ[0],
					 tetX[2] - tetX[0], tetY[2] - tetY[0], tetZ[2] - tetZ[0],
					 tetX[3] - tetX[0], tetY[3] - tetY[0], tetZ[3] - tetZ[0]};

	double V = 1 / DeterminantThree(det);

	double z41 = tetZ[3] - tetZ[0];
	double y34 = tetY[2] - tetY[3];
	double z34 = tetZ[2] - tetZ[3];
	double y41 = tetY[3] - tetY[0];
	double a11 = (z41 * y34 - z34 * y41) * V;

	double x41 = tetX[3] - tetX[0];
	double x34 = tetX[2] - tetX[3];
	double a12 = (x41 * z34 - x34 * z41) * V;

	double a13 = (y41 * x34 - y34 * x41) * V;

	coordinates[1] = a11 * X + a12 * Y + a13 * Z;

	double y12 = tetY[0] - tetY[1];
	double z12 = tetZ[0] - tetZ[1];
	double a21 = (z41 * y12 - z12 * y41) * V;

	double x12 = tetX[0] - tetX[1];
	double a22 = (x41 * z12 - x12 * z41) * V;

	double a23 = (y41 * x12 - y12 * x41) * V;

	coordinates[2] = a21 * X + a22 * Y + a23 * Z;

	double z23 = tetZ[1] - tetZ[2];
	double y23 = tetY[1] - tetY[2];
	double a31 = (z23 * y12 - z12 * y23) * V;

	double x23 = tetX[1] - tetX[2];
	double a32 = (x23 * z12 - x12 * z23) * V;

	double a33 = (y23 * x12 - y12 * x23) * V;

	coordinates[3] = a31 * X + a32 * Y + a33 * Z;

	coordinates[0] = 1 - coordinates[1] - coordinates[2] - coordinates[3];
}

__device__ inline int gFindCell(double *particle, int *connectivities, int *links,
					 double *vertexPositions,
					 double epsilon, int guess, double *coordinates) {
	double tetX[4], tetY[4], tetZ[4];

	while (true) {
		for (int i = 0; i < 4; i++) {
			int pointID = connectivities[(guess << 2) | i];

			tetX[i] = vertexPositions[pointID * 3];
			tetY[i] = vertexPositions[pointID * 3 + 1];
			tetZ[i] = vertexPositions[pointID * 3 + 2];
		}

		CalculateNaturalCoordinates(particle[0], particle[1], particle[2], tetX, tetY, tetZ, coordinates);

		int index = 0;

		for (int i = 1; i < 4; i++)
			if (coordinates[i] < coordinates[index]) index = i;

		if (index >= 0 && index <= 3)
			if (coordinates[index] >= -epsilon) break;

		guess = links[(guess << 2) | index];
		
		if (guess == -1) break;
	}

	return guess;
}

__device__ inline int FindCell(double *particle, int *connectivities, int *links, double *vertexPositions,
					double epsilon, int guess, double *coordinates) {
	double tetX[4], tetY[4], tetZ[4];
	while (true) {
		for (int i = 0; i < 4; i++) {
			int pointID = connectivities[(guess << 2) | i];
			tetX[i] = vertexPositions[pointID * 3];
			tetY[i] = vertexPositions[pointID * 3 + 1];
			tetZ[i] = vertexPositions[pointID * 3 + 2];
		}
		CalculateNaturalCoordinates(particle[0], particle[1], particle[2], tetX, tetY, tetZ, coordinates);
		int index = 0;
		for (int i = 1; i < 4; i++)
			if (coordinates[i] < coordinates[index]) index = i;
		if (coordinates[index] >= -epsilon) break;
		guess = links[(guess << 2) | index];
		if (guess == -1) break;
	}
	return guess;
}

__global__ void BlockedTracing(double *globalVertexPositions,
							   double *globalStartVelocities,
							   double *globalEndVelocities,
							   int *globalTetrahedralConnectivities,
							   int *globalTetrahedralLinks,

							   int *startOffsetInCell,
							   int *startOffsetInPoint,

							   int *startOffsetInCellForBig,
							   int *startOffsetInPointForBig,
							   double *vertexPositionsForBig,
							   double *startVelocitiesForBig,
							   double *endVelocitiesForBig,

							   bool *canFitInSharedMemory,

							   int *blockedLocalConnectivities,
							   int *blockedLocalLinks,
							   int *blockedGlobalCellIDs,
							   int *blockedGlobalPointIDs,

							   int *activeBlockList, // Map active block ID to interesting block ID

							   int *stage,
							   double *lastPosition,
							   double *k1,
							   double *k2,
							   double *k3,
							   double *pastTimes,
							   int *startOffsetInParticle,
							   int *blockedActiveParticleIDList,
							   int *blockedCellLocationList,

							   /// shared memory size
							   //int sharedMemoryBytes,
							 
							   double startTime, double endTime, double timeStep,
							   double epsilon,
							 
							   int *squeezedStage,
							   double *squeezedLastPosition,
							   double *squeezedK1,
							   double *squeezedK2,
							   double *squeezedK3,
							   int *squeezedExitCells
							 ) {
	//printf("startTime = %lf, endTime = %lf, timeStep = %lf\n", startTime, endTime, timeStep);

	//extern __shared__ char sharedMemory[];
	//__shared__ char sharedMemory[16384];
	__shared__ char sharedMemory[8192];
	//char *sharedMemory;

	int globalID = blockIdx.x * blockDim.x + threadIdx.x;

	//printf("I am in block %d, with thread id %d.\n", blockIdx.x, threadIdx.x);

	// Get work group ID, which is equal to active block ID
	int activeBlockID = blockIdx.x;
	
	// Get number of threads in a work group
	int numOfThreads = blockDim.x;

	// Get local thread ID
	int localID = threadIdx.x;

	// Get interesting block ID of the current active block ID
	int interestingBlockID = activeBlockList[activeBlockID];

	// Declare some arrays
	double *vertexPositions;
	double *startVelocities;
	double *endVelocities;
	int *connectivities;
	int *links;

	double *gVertexPositions;
	double *gStartVelocities;
	double *gEndVelocities;
	int *gConnectivities;
	int *gLinks;

	bool canFit = canFitInSharedMemory[interestingBlockID];

	int startCell = startOffsetInCell[interestingBlockID];
	int startPoint = startOffsetInPoint[interestingBlockID];

	int numOfCells = startOffsetInCell[interestingBlockID + 1] - startCell;
	int numOfPoints = startOffsetInPoint[interestingBlockID + 1] - startPoint;

	int startCellForBig = startOffsetInCellForBig[interestingBlockID];
	int startPointForBig = startOffsetInPointForBig[interestingBlockID];

	if (canFit) { // This branch fills in the shared memory
		// Initialize vertexPositions, startVelocities and endVelocities
		vertexPositions = (double *)sharedMemory;
		startVelocities = vertexPositions + numOfPoints * 3;
		endVelocities = startVelocities + numOfPoints * 3;

		// Initialize connectivities and links
		connectivities = (int *)(endVelocities + numOfPoints * 3);
		links = connectivities + (numOfCells << 2);
	} else { // This branch fills in the global memory
		// Initialize vertexPositions, startVelocities and endVelocities
		gVertexPositions = vertexPositionsForBig + startPointForBig * 3;
		gStartVelocities = startVelocitiesForBig + startPointForBig * 3;
		gEndVelocities = endVelocitiesForBig + startPointForBig * 3;

		// Initialize connectivities and links
		gConnectivities = blockedLocalConnectivities + (startCell << 2);
		gLinks = blockedLocalLinks + (startCell << 2);
	}

	for (int i = localID; i < numOfPoints * 3; i += numOfThreads) {
		int localPointID = i / 3;
		int dimensionID = i % 3;
		int globalPointID = blockedGlobalPointIDs[startPoint + localPointID];

		if (canFit) {
			vertexPositions[i] = globalVertexPositions[globalPointID * 3 + dimensionID];
			startVelocities[i] = globalStartVelocities[globalPointID * 3 + dimensionID];
			endVelocities[i] = globalEndVelocities[globalPointID * 3 + dimensionID];
		} else {
			/*gVertexPositions[i] = gliobalVertexPositions[globalPointID * 3 + dimensionID];
			gStartVelocities[i] = globalStartVelocities[globalPointID * 3 + dimensionID];
			gEndVelocities[i] = globalEndVelocities[globalPointID * 3 + dimensionID];*/
		}
	}

	if (canFit)
		for (int i = localID; i < (numOfCells << 2); i += numOfThreads) {
			connectivities[i] = *(blockedLocalConnectivities + (startCell << 2) + i);
			links[i] = *(blockedLocalLinks + (startCell << 2) + i);
		}

	__syncthreads();
	
	int numOfActiveParticles = startOffsetInParticle[activeBlockID + 1] - startOffsetInParticle[activeBlockID];

	for (int idx = localID; idx < numOfActiveParticles; idx += numOfThreads) {
		//printf("blk = %d, trd = %d, idx = %d\n", blockIdx.x, threadIdx.x, idx);

		// activeParticleID here means the initial active particle ID
		int arrayIdx = startOffsetInParticle[activeBlockID] + idx;
		int activeParticleID = blockedActiveParticleIDList[arrayIdx];

		/// DEBUG ///
		bool debug = activeParticleID == 1269494;

		// Initialize the particle status
		int currStage = stage[activeParticleID];
		int currCell = blockedCellLocationList[startOffsetInParticle[activeBlockID] + idx];

		double currTime = pastTimes[activeParticleID];

		double currLastPosition[3];
		currLastPosition[0] = lastPosition[activeParticleID * 3];
		currLastPosition[1] = lastPosition[activeParticleID * 3 + 1];
		currLastPosition[2] = lastPosition[activeParticleID * 3 + 2];
		double currK1[3], currK2[3], currK3[3], currK4[3];
		if (currStage > 0) {
			currK1[0] = k1[activeParticleID * 3];
			currK1[1] = k1[activeParticleID * 3 + 1];
			currK1[2] = k1[activeParticleID * 3 + 2];
		}
		if (currStage > 1) {
			currK2[0] = k2[activeParticleID * 3];
			currK2[1] = k2[activeParticleID * 3 + 1];
			currK2[2] = k2[activeParticleID * 3 + 2];
		}
		if (currStage > 2) {
			currK3[0] = k3[activeParticleID * 3];
			currK3[1] = k3[activeParticleID * 3 + 1];
			currK3[2] = k3[activeParticleID * 3 + 2];
		}

		int cnt = 0;

		// At least one loop is executed.
		while (true) {

			/// DEBUG ///
			cnt++;

			double placeOfInterest[3];
			placeOfInterest[0] = currLastPosition[0];
			placeOfInterest[1] = currLastPosition[1];
			placeOfInterest[2] = currLastPosition[2];
			switch (currStage) {
			case 1: {
				placeOfInterest[0] += 0.5 * currK1[0];
				placeOfInterest[1] += 0.5 * currK1[1];
				placeOfInterest[2] += 0.5 * currK1[2];
					} break;
			case 2: {
				placeOfInterest[0] += 0.5 * currK2[0];
				placeOfInterest[1] += 0.5 * currK2[1];
				placeOfInterest[2] += 0.5 * currK2[2];
					} break;
			case 3: {
				placeOfInterest[0] += currK3[0];
				placeOfInterest[1] += currK3[1];
				placeOfInterest[2] += currK3[2];
					} break;
			}

			double coordinates[4];

			int nextCell;
			
			if (canFit)
				nextCell = FindCell(placeOfInterest, connectivities, links, vertexPositions, epsilon, currCell, coordinates);
			else /// DEBUG ///
				nextCell = gFindCell(placeOfInterest, gConnectivities, gLinks, gVertexPositions, epsilon, currCell, coordinates);

			if (nextCell == -1 || currTime >= endTime) {
				// Find the next cell globally
				int globalCellID = blockedGlobalCellIDs[startCell + currCell];
				int nextGlobalCell;
			
				if (nextCell != -1)
					nextGlobalCell = blockedGlobalCellIDs[startCell + nextCell];
				else
					nextGlobalCell = gFindCell(placeOfInterest, globalTetrahedralConnectivities, globalTetrahedralLinks,
											   globalVertexPositions, epsilon, globalCellID, coordinates);

				if (currTime >= endTime && nextGlobalCell != -1) nextGlobalCell = -2 - nextGlobalCell;

				pastTimes[activeParticleID] = currTime;

				stage[activeParticleID] = currStage;

				lastPosition[activeParticleID * 3] = currLastPosition[0];
				lastPosition[activeParticleID * 3 + 1] = currLastPosition[1];
				lastPosition[activeParticleID * 3 + 2] = currLastPosition[2];
		
				if (currStage > 0) {
					k1[activeParticleID * 3] = currK1[0];
					k1[activeParticleID * 3 + 1] = currK1[1];
					k1[activeParticleID * 3 + 2] = currK1[2];
				}
				if (currStage > 1) {
					k2[activeParticleID * 3] = currK2[0];
					k2[activeParticleID * 3 + 1] = currK2[1];
					k2[activeParticleID * 3 + 2] = currK2[2];
				}
				if (currStage > 2) {
					k3[activeParticleID * 3] = currK3[0];
					k3[activeParticleID * 3 + 1] = currK3[1];
					k3[activeParticleID * 3 + 2] = currK3[2];
				}

				// Write squeezed arrays
				squeezedStage[arrayIdx] = currStage;
				squeezedExitCells[arrayIdx] = nextGlobalCell;

				squeezedLastPosition[arrayIdx * 3] = currLastPosition[0];
				squeezedLastPosition[arrayIdx * 3 + 1] = currLastPosition[1];
				squeezedLastPosition[arrayIdx * 3 + 2] = currLastPosition[2];
		
				if (currStage > 0) {
					squeezedK1[arrayIdx * 3] = currK1[0];
					squeezedK1[arrayIdx * 3 + 1] = currK1[1];
					squeezedK1[arrayIdx * 3 + 2] = currK1[2];
				}
				if (currStage > 1) {
					squeezedK2[arrayIdx * 3] = currK2[0];
					squeezedK2[arrayIdx * 3 + 1] = currK2[1];
					squeezedK2[arrayIdx * 3 + 2] = currK2[2];
				}
				if (currStage > 2) {
					squeezedK3[arrayIdx * 3] = currK3[0];
					squeezedK3[arrayIdx * 3 + 1] = currK3[1];
					squeezedK3[arrayIdx * 3 + 2] = currK3[2];
				}

				break;
			}

			currCell = nextCell;

			double exactTime = currTime;
			switch (currStage) {
			case 0: break;
			case 1:
			case 2: exactTime += timeStep * 0.5; break;
			case 3: exactTime += timeStep; break;
			}

			double alpha = (endTime - exactTime) / (endTime - startTime);
			double beta = 1 - alpha;

			double vecX[4], vecY[4], vecZ[4];

			for (int i = 0; i < 4; i++)
				if (canFit) {
					int pointID = connectivities[(nextCell << 2) | i];
					vecX[i] = startVelocities[pointID * 3] * alpha + endVelocities[pointID * 3] * beta;
					vecY[i] = startVelocities[pointID * 3 + 1] * alpha + endVelocities[pointID * 3 + 1] * beta;
					vecZ[i] = startVelocities[pointID * 3 + 2] * alpha + endVelocities[pointID * 3 + 2] * beta;
				} else {
					int pointID = gConnectivities[(nextCell << 2) | i];
					vecX[i] = gStartVelocities[pointID * 3] * alpha + gEndVelocities[pointID * 3] * beta;
					vecY[i] = gStartVelocities[pointID * 3 + 1] * alpha + gEndVelocities[pointID * 3 + 1] * beta;
					vecZ[i] = gStartVelocities[pointID * 3 + 2] * alpha + gEndVelocities[pointID * 3 + 2] * beta;
				}

			double *currK;
			switch (currStage) {
			case 0: currK = currK1; break;
			case 1: currK = currK2; break;
			case 2: currK = currK3; break;
			case 3: currK = currK4; break;
			}

			currK[0] = currK[1] = currK[2] = 0;

			for (int i = 0; i < 4; i++) {
				currK[0] += vecX[i] * coordinates[i];
				currK[1] += vecY[i] * coordinates[i];
				currK[2] += vecZ[i] * coordinates[i];
			}

			///// DEBUG ///
			//if (debug && currStage == 0) {
			//	printf("vec = %lf %lf %lf\n", currK[0], currK[1], currK[2]);
			//}
			//if (debug && currStage == 0 && currCell != -1 && blockedGlobalCellIDs[startCell + currCell] == 161660) {
			//	int pointID = connectivities[nextCell << 2];
			//	printf("startVec[0] = %lf %lf %lf, endVec[0] = %lf %lf %lf\n", startVelocities[pointID * 3], startVelocities[pointID * 3 + 1], startVelocities[pointID * 3 + 2],
			//																   endVelocities[pointID * 3], endVelocities[pointID * 3 + 1], endVelocities[pointID * 3 + 2]);
			//	printf("coordinates:");
			//	for (int i = 0; i < 4; i++)
			//		printf(" %lf", coordinates[i]);
			//	printf("\n");

			//	for (int i = 0; i < 4; i++)
			//		printf("point %d: %lf %lf %lf\n", i, vecX[i], vecY[i], vecZ[i]);
			//}

			currK[0] *= timeStep;
			currK[1] *= timeStep;
			currK[2] *= timeStep;

			if (currStage == 3) {
				currTime += timeStep;

				for (int i = 0; i < 3; i++)
					currLastPosition[i] += (currK1[i] + 2 * currK2[i] + 2 * currK3[i] + currK4[i]) / 6;

				currStage = 0;
			} else
				currStage++;
		}
	}
}	
