/*****************************************************
File		:	lcsBlockedTracingOfRK4.cu
Author		:	Mingcheng Chen
Last Update	:	March 20th, 2013
******************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

//#define USE_CACHE

/*
__device__ inline double DeterminantThree(double *a) {
	// a[0] a[1] a[2]
	// a[3] a[4] a[5]
	// a[6] a[7] a[8]
	//return a[0] * a[4] * a[8] + a[1] * a[5] * a[6] + a[2] * a[3] * a[7] -
	//       a[0] * a[5] * a[7] - a[1] * a[3] * a[8] - a[2] * a[4] * a[6];
	return a[0] * (a[4] * a[8] - a[5] * a[7]) + a[1] * (a[5] * a[6] - a[3] * a[8]) + a[2] * (a[3] * a[7] - a[4] * a[6]);
}
*/

__constant__ void *pointers[25];
__constant__ double timeStep, epsilon;
__constant__ int sharedMemorySize, multiple;
//__constant__ double doubleValues[4];

__device__ int FindCell(double *particle, int *connectivities, int *links, double *vertexPositions,
			double epsilon, int guess, double *coordinates, double *tetX, double *tetY, double *tetZ) {
	//double tetX[4], tetY[4], tetZ[4];
	int index, pointID;
	double X, Y, Z, V;
	double z41, y34, z34, y41, /*a11, */x41, x34, /*a12, a13, */y12, z12, /*a21, */x12, /*a22, a23, */z23, y23, /*a31, */x23/*, a32, a33*/;

	while (true) {
		for (index = 0; index < 4; index++) {
			pointID = connectivities[(guess << 2) | index] * 3;

			tetX[index] = vertexPositions[pointID /** 3*/];
			tetY[index] = vertexPositions[pointID /** 3*/ + 1];
			tetZ[index] = vertexPositions[pointID /** 3*/ + 2];
		}

		//CalculateNaturalCoordinates(particle[0], particle[1], particle[2], tetX, tetY, tetZ, coordinates);

		X = particle[0] - tetX[0];
		Y = particle[1] - tetY[0];
		Z = particle[2] - tetZ[0];

		tetX[1] -= tetX[0];
		tetX[2] -= tetX[0];
		tetX[3] -= tetX[0];

		tetY[1] -= tetY[0];
		tetY[2] -= tetY[0];
		tetY[3] -= tetY[0];

		tetZ[1] -= tetZ[0];
		tetZ[2] -= tetZ[0];
		tetZ[3] -= tetZ[0];

		//double a[9] = {tetX[1] - tetX[0], tetY[1] - tetY[0], tetZ[1] - tetZ[0],
		//		tetX[2] - tetX[0], tetY[2] - tetY[0], tetZ[2] - tetZ[0],
		//		tetX[3] - tetX[0], tetY[3] - tetY[0], tetZ[3] - tetZ[0]};

		//V = 1.0 / (a[0] * (a[4] * a[8] - a[5] * a[7]) + a[1] * (a[5] * a[6] - a[3] * a[8]) + a[2] * (a[3] * a[7] - a[4] * a[6]));

		V = 1.0 / (tetX[1] * (tetY[2] * tetZ[3] - tetZ[2] * tetY[3]) + 
			   tetY[1] * (tetZ[2] * tetX[3] - tetX[2] * tetZ[3]) +
			   tetZ[1] * (tetX[2] * tetY[3] - tetY[2] * tetX[3]));

		z41 = tetZ[3];// - tetZ[0];
		y34 = tetY[2] - tetY[3];
		z34 = tetZ[2] - tetZ[3];
		y41 = tetY[3];// - tetY[0];
		//a11 = z41 * y34 - z34 * y41;

		x41 = tetX[3];// - tetX[0];
		x34 = tetX[2] - tetX[3];
		//a12 = x41 * z34 - x34 * z41;

		//a13 = y41 * x34 - y34 * x41;

		//coordinates[1] = (a11 * X + a12 * Y + a13 * Z) * V;
		coordinates[1] = ((z41 * y34 - z34 * y41) * X + (x41 * z34 - x34 * z41) * Y + (y41 * x34 - y34 * x41) * Z) * V;

		y12 = -tetY[1]; // tetY[0] - tetY[1];
		z12 = -tetZ[1]; // tetZ[0] - tetZ[1];
		//a21 = z41 * y12 - z12 * y41;

		x12 = -tetX[1]; // tetX[0] - tetX[1];
		//a22 = x41 * z12 - x12 * z41;

		//a23 = y41 * x12 - y12 * x41;

		//coordinates[2] = (a21 * X + a22 * Y + a23 * Z) * V;
		coordinates[2] = ((z41 * y12 - z12 * y41) * X + (x41 * z12 - x12 * z41) * Y + (y41 * x12 - y12 * x41) * Z) * V;
		index = coordinates[2] < coordinates[1] ? 2 : 1;

		z23 = tetZ[1] - tetZ[2];
		y23 = tetY[1] - tetY[2];
		//a31 = z23 * y12 - z12 * y23;

		x23 = tetX[1] - tetX[2];
		//a32 = x23 * z12 - x12 * z23;

		//a33 = y23 * x12 - y12 * x23;

		if ((coordinates[3] = ((z23 * y12 - z12 * y23) * X + (x23 * z12 - x12 * z23) * Y + (y23 * x12 - y12 * x23) * Z) * V) < coordinates[index]) index = 3;

		if ((coordinates[0] = 1.0 - coordinates[1] - coordinates[2] - coordinates[3]) < coordinates[index]) index = 0;

		if (coordinates[index] >= -epsilon) break;

		guess = links[(guess << 2) | index];
		
		if (guess == -1) break;
	}

	return guess;
}

__global__ void BlockedTracingKernelOfRK4(/*double *globalVertexPositions,
					int *globalTetrahedralConnectivities,
					int *globalTetrahedralLinks,

					int *startOffsetInCell,
					int *startOffsetInPoint,

					double *vertexPositionsForBig,
					double *startVelocitiesForBig,
					double *endVelocitiesForBig,

					int *blockedLocalConnectivities,
					int *blockedLocalLinks,
					int *blockedGlobalCellIDs,

					int *activeBlockList, // Map active block ID to interesting block ID

					int *blockOfGroups,
					int *offsetInBlocks,

					int *stage,
					double *lastPosition,
					double *k1,
					double *k2,
					double *k3,
					double *pastTimes,

					double *placesOfInterest,

					int *startOffsetInParticle,
					int *blockedActiveParticleIDList,
					int *cellLocations,

					int *exitCells,
*/
					double startTime, double endTime, double timeStep, double epsilon //, int sharedMemorySize, int multiple*/
					) {
/*
	cudaError_t err = cudaMemcpyToSymbol(pointers, &globalVertexPositions, sizeOfPointer, 0, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &globalTetrahedralConnectivities, sizeOfPointer, sizeOfPointer, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &globalTetrahedralLinks, sizeOfPointer, sizeOfPointer * 2, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInCell, sizeOfPointer, sizeOfPointer * 3, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInPoint, sizeOfPointer, sizeOfPointer * 4, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &vertexPositionsForBig, sizeOfPointer, sizeOfPointer * 5, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startVelocitiesForBig, sizeOfPointer, sizeOfPointer * 6, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &endVelocitiesForBig, sizeOfPointer, sizeOfPointer * 7, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedLocalConnectivities, sizeOfPointer, sizeOfPointer * 8, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedLocalLinks, sizeOfPointer, sizeOfPointer * 9, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedGlobalCellIDs, sizeOfPointer, sizeOfPointer * 10, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &activeBlockList, sizeOfPointer, sizeOfPointer * 11, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockOfGroups, sizeOfPointer, sizeOfPointer * 12, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &offsetInBlocks, sizeOfPointer, sizeOfPointer * 13, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &stage, sizeOfPointer, sizeOfPointer * 14, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &lastPosition, sizeOfPointer, sizeOfPointer * 15, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k1, sizeOfPointer, sizeOfPointer * 16, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k2, sizeOfPointer, sizeOfPointer * 17, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k3, sizeOfPointer, sizeOfPointer * 18, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &pastTimes, sizeOfPointer * 19, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &placesOfInterest, sizeOfPointer, sizeOfPointer * 20, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInParticle, sizeOfPointer, sizeOfPointer * 21, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedActiveParticleIDList, sizeOfPointer, sizeOfPointer * 22, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &cellLocations, sizeOfPointer, sizeOfPointer * 23, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &exitCells, sizeOfPointer, sizeOfPointer * 24, cudaMemcpyHostToDevice) |
*/

#ifndef USE_CACHE
	__shared__ extern char sharedMemory[];
#endif

	//char *sharedMemory;

	// Get work group ID
	//int groupID = blockIdx.x;
	
	// Get number of threads in a work group
	//int numOfThreads = blockDim.x;

	// Get local thread ID
	//int localID = threadIdx.x;

	// Get active block ID
	int activeBlockID = ((int *)pointers[12])[blockIdx.x/*groupID*/];
	//int activeBlockID = blockOfGroups[groupID];

	// Get interesting block ID of the work group
	int i, idx;
	i = ((int *)pointers[11])[activeBlockID];
	//int interestingBlockID = ((int *)pointers[11])[activeBlockID];
	//int interestingBlockID = activeBlockList[activeBlockID];

	// Declare some arrays
	double *vertexPositions;
	double *startVelocities;
	double *endVelocities;
	int *connectivities;
	int *links;

	int startCell = ((int *)pointers[3])[i/*interestingBlockID*/];
	int offset/*int startPoint*/ = ((int *)pointers[4])[i/*interestingBlockID*/];
	idx = offset * 3;
	//int startCell = startOffsetInCell[interestingBlockID];
	//int startPoint = startOffsetInPoint[interestingBlockID];

	int numOfCells = ((int *)pointers[3])[i/*interestingBlockID*/ + 1] - startCell;
	int numOfPoints = ((int *)pointers[4])[i/*interestingBlockID*/ + 1] - offset/*startPoint*/;

	//startPoint *= 3;
	//int numOfCells = startOffsetInCell[interestingBlockID + 1] - startCell;
	//int numOfPoints = startOffsetInPoint[interestingBlockID + 1] - startPoint;

	// Assuming int is 4 bytes and double is 8 bytes
	//	 localNumOfCells * sizeof(int) * 4 +		// this->localConnectivities
	//	 localNumOfCells * sizeof(int) * 4 +		// this->localLinks
	//	 localNumOfPoints * sizeof(double) * 3 +	// point positions
	//	 localNumOfPoints * sizeof(double) * 3 * 2;	// point velocities (start and end)

#ifndef USE_CACHE
	if (((numOfCells << 5) + ((numOfPoints * 9) << 3)) <= sharedMemorySize) { // This branch fills in the shared memory
		// Initialize vertexPositions, startVelocities and endVelocities
		vertexPositions = (double *)sharedMemory;
		startVelocities = vertexPositions + numOfPoints * 3;
		endVelocities = startVelocities + numOfPoints * 3;

		// Initialize connectivities and links
		connectivities = (int *)(endVelocities + numOfPoints * 3);
		links = connectivities + (numOfCells << 2);

		for (/*int*/ i = threadIdx.x/*localID*/; i < numOfPoints * 3; i += blockDim.x/*numOfThreads*/) {
			vertexPositions[i] = ((double *)pointers[5])[idx/*startPoint * 3*/ + i];
			startVelocities[i] = ((double *)pointers[6])[idx/*startPoint * 3*/ + i];
			endVelocities[i] = ((double *)pointers[7])[idx/*startPoint * 3*/ + i];
			//vertexPositions[i] = vertexPositionsForBig[startPoint * 3 + i];
			//startVelocities[i] = startVelocitiesForBig[startPoint * 3 + i];
			//endVelocities[i] = endVelocitiesForBig[startPoint * 3 + i];
		}

		for (/*int*/ i = threadIdx.x/*localID*/; i < (numOfCells << 2); i += blockDim.x/*numOfThreads*/) {
			connectivities[i] = ((int *)pointers[8])[(startCell << 2) + i];
			links[i] = ((int *)pointers[9])[(startCell << 2) + i];
			//connectivities[i] = blockedLocalConnectivities[(startCell << 2) + i];
			//links[i] = blockedLocalLinks[(startCell << 2) + i];
		}

		__syncthreads();
	} else
#endif
	{ // This branch fills in the global memory
		// Initialize vertexPositions, startVelocities and endVelocities
		vertexPositions = (double *)pointers[5] + idx/*startPoint * 3*/;
		startVelocities = (double *)pointers[6] + idx/*startPoint * 3*/;
		endVelocities = (double *)pointers[7] + idx/*startPoint * 3*/;
		//vertexPositions = vertexPositionsForBig + startPoint * 3;
		//startVelocities = startVelocitiesForBig + startPoint * 3;
		//endVelocities = endVelocitiesForBig + startPoint * 3;

		// Initialize connectivities and links
		connectivities = (int *)pointers[8] + (startCell << 2);
		links = (int *)pointers[9] + (startCell << 2);
		//connectivities = blockedLocalConnectivities + (startCell << 2);
		//links = blockedLocalLinks + (startCell << 2);
	}

	//__syncthreads();

	int numOfActiveParticles = ((int *)pointers[21])[activeBlockID + 1] - ((int *)pointers[21])[activeBlockID];
	/*int*/ offset = ((int *)pointers[13])[blockIdx.x/*groupID*/] * blockDim.x/*numOfThreads*/ * multiple;
	//int numOfActiveParticles = startOffsetInParticle[activeBlockID + 1] - startOffsetInParticle[activeBlockID];
	//int offset = offsetInBlocks[groupID] * numOfThreads * multiple;

	int activeParticleID, currStage, currCell, nextCell;
	//int nextGlobalCell;

	double currTime;
	double currLastPosition[3], currK[3], currNX[3]; //currK2[3], currK3[3], currK4[3];
	double placeOfInterest[3];
	double coordinates[4];
	double alpha, beta;

	//double vecX[4], vecY[4], vecZ[4];
	double vec[12];
	//double *currK;

	for (idx = threadIdx.x/*localID*/; idx < blockDim.x/*numOfThreads*/ * multiple; idx += blockDim.x/*numOfThreads*/) {
		//int arrayIdx = offsetInBlocks[groupID] * numOfThreads + localID;
		activeParticleID = offset + idx;
		//int arrayIdx = offset + idx;

		//if (arrayIdx < numOfActiveParticles) {
		if (activeParticleID < numOfActiveParticles) {
			// activeParticleID here means the initial active particle ID
			//arrayIdx += ((int *)pointers[21])[activeBlockID];
			//int activeParticleID = ((int *)pointers[22])[arrayIdx];
			activeParticleID = ((int *)pointers[22])[activeParticleID + ((int *)pointers[21])[activeBlockID]];
			//arrayIdx += startOffsetInParticle[activeBlockID];
			//int activeParticleID = blockedActiveParticleIDList[arrayIdx];

			// Initialize the particle status
			/*int*/ currStage = ((int *)pointers[14])[activeParticleID];
			/*int*/ currCell = ((int *)pointers[23])[activeParticleID];
			//int currStage = stage[activeParticleID];
			//int currCell = cellLocations[activeParticleID];

			/*double*/ currTime = ((double *)pointers[19])[activeParticleID];
			//double currTime = pastTimes[activeParticleID];

			/*double currLastPosition[3];*/
			currLastPosition[0] = ((double *)pointers[15])[activeParticleID * 3];
			currLastPosition[1] = ((double *)pointers[15])[activeParticleID * 3 + 1];
			currLastPosition[2] = ((double *)pointers[15])[activeParticleID * 3 + 2];
			//currLastPosition[0] = lastPosition[activeParticleID * 3];
			//currLastPosition[1] = lastPosition[activeParticleID * 3 + 1];
			//currLastPosition[2] = lastPosition[activeParticleID * 3 + 2];
			/*double currK1[3], currK2[3], currK3[3], currK4[3];*/

			/*
			if (currStage > 0) {
				currK1[0] = ((double *)pointers[16])[activeParticleID * 3];
				currK1[1] = ((double *)pointers[16])[activeParticleID * 3 + 1];
				currK1[2] = ((double *)pointers[16])[activeParticleID * 3 + 2];
				//currK1[0] = k1[activeParticleID * 3];
				//currK1[1] = k1[activeParticleID * 3 + 1];
				//currK1[2] = k1[activeParticleID * 3 + 2];
			}
			if (currStage > 1) {
				currK2[0] = ((double *)pointers[17])[activeParticleID * 3];
				currK2[1] = ((double *)pointers[17])[activeParticleID * 3 + 1];
				currK2[2] = ((double *)pointers[17])[activeParticleID * 3 + 2];
				//currK2[0] = k2[activeParticleID * 3];
				//currK2[1] = k2[activeParticleID * 3 + 1];
				//currK2[2] = k2[activeParticleID * 3 + 2];
			}
			if (currStage > 2) {
				currK3[0] = ((double *)pointers[18])[activeParticleID * 3];
				currK3[1] = ((double *)pointers[18])[activeParticleID * 3 + 1];
				currK3[2] = ((double *)pointers[18])[activeParticleID * 3 + 2];
				//currK3[0] = k3[activeParticleID * 3];
				//currK3[1] = k3[activeParticleID * 3 + 1];
				//currK3[2] = k3[activeParticleID * 3 + 2];
			}
			*/
			currK[0] = ((double *)pointers[16])[activeParticleID * 3];
			currK[1] = ((double *)pointers[16])[activeParticleID * 3 + 1];
			currK[2] = ((double *)pointers[16])[activeParticleID * 3 + 2];

			currNX[0] = ((double *)pointers[17])[activeParticleID * 3];
			currNX[1] = ((double *)pointers[17])[activeParticleID * 3 + 1];
			currNX[2] = ((double *)pointers[17])[activeParticleID * 3 + 2];

			// At least one loop is executed.
			while (true) {
				/*double placeOfInterest[3];*/
				placeOfInterest[0] = currLastPosition[0];
				placeOfInterest[1] = currLastPosition[1];
				placeOfInterest[2] = currLastPosition[2];

				switch (currStage) {
				case 1:
				case 2: {
					placeOfInterest[0] += 0.5 * currK[0];
					placeOfInterest[1] += 0.5 * currK[1];
					placeOfInterest[2] += 0.5 * currK[2];
					} break;
				case 3: {
					placeOfInterest[0] += currK[0];
					placeOfInterest[1] += currK[1];
					placeOfInterest[2] += currK[2];
					} break;
				}

				/*double coordinates[4];*/

				//int nextCell;
				nextCell = FindCell(placeOfInterest, connectivities, links, vertexPositions, /*doubleValues[3]*/epsilon, currCell, coordinates, vec, vec + 4, vec + 8);

				if (nextCell == -1 || currTime >= /*doubleValues[1]*/endTime) {
					// Find the next cell globally
					//int globalCellID = ((int *)pointers[10])[startCell + currCell];
					//int globalCellID = blockedGlobalCellIDs[startCell + currCell];
					//int nextGlobalCell;
				
					if (nextCell != -1)
						nextCell/*nextGlobalCell*/ = ((int *)pointers[10])[startCell + nextCell];
						//nextGlobalCell = blockedGlobalCellIDs[startCell + nextCell];
					else
						nextCell/*nextGlobalCell*/ = FindCell(placeOfInterest, (int *)pointers[1], (int *)pointers[2], (double *)pointers[0],
									/*doubleValues[3]*/epsilon, /*globalCellID*/((int *)pointers[10])[startCell + currCell], coordinates, vec, vec + 4, vec + 8);
						//nextGlobalCell = FindCell(placeOfInterest, globalTetrahedralConnectivities,
						//			globalTetrahedralLinks, globalVertexPositions,
						//			epsilon, globalCellID, coordinates);

					if (currTime >= /*doubleValues[1]*/endTime && nextCell/*nextGlobalCell*/ != -1) nextCell = -2 - nextCell;//nextGlobalCell = -2 - nextGlobalCell;

					((double *)pointers[19])[activeParticleID] = currTime;
					//pastTimes[activeParticleID] = currTime;

					((int *)pointers[14])[activeParticleID] = currStage;
					//stage[activeParticleID] = currStage;

					((double *)pointers[15])[activeParticleID * 3] = currLastPosition[0];
					((double *)pointers[15])[activeParticleID * 3 + 1] = currLastPosition[1];
					((double *)pointers[15])[activeParticleID * 3 + 2] = currLastPosition[2];
					//lastPosition[activeParticleID * 3] = currLastPosition[0];
					//lastPosition[activeParticleID * 3 + 1] = currLastPosition[1];
					//lastPosition[activeParticleID * 3 + 2] = currLastPosition[2];

					((double *)pointers[20])[activeParticleID * 3] = placeOfInterest[0];
					((double *)pointers[20])[activeParticleID * 3 + 1] = placeOfInterest[1];
					((double *)pointers[20])[activeParticleID * 3 + 2] = placeOfInterest[2];
					//placesOfInterest[activeParticleID * 3] = placeOfInterest[0];
					//placesOfInterest[activeParticleID * 3 + 1] = placeOfInterest[1];
					//placesOfInterest[activeParticleID * 3 + 2] = placeOfInterest[2];

					((int *)pointers[24])[activeParticleID] = nextCell;//nextGlobalCell;
					//exitCells[activeParticleID] = nextGlobalCell;

					/*
					if (currStage > 0) { // currStage > 0
						((double *)pointers[16])[activeParticleID * 3] = currK1[0];
						((double *)pointers[16])[activeParticleID * 3 + 1] = currK1[1];
						((double *)pointers[16])[activeParticleID * 3 + 2] = currK1[2];
						//k1[activeParticleID * 3] = currK1[0];
						//k1[activeParticleID * 3 + 1] = currK1[1];
						//k1[activeParticleID * 3 + 2] = currK1[2];
					}	
					if (currStage > 1) { // currStage > 1
						((double *)pointers[17])[activeParticleID * 3] = currK2[0];
						((double *)pointers[17])[activeParticleID * 3 + 1] = currK2[1];
						((double *)pointers[17])[activeParticleID * 3 + 2] = currK2[2];
						//k2[activeParticleID * 3] = currK2[0];
						//k2[activeParticleID * 3 + 1] = currK2[1];
						//k2[activeParticleID * 3 + 2] = currK2[2];
					}	
					if (currStage > 2) { // currStage > 2
						((double *)pointers[18])[activeParticleID * 3] = currK3[0];
						((double *)pointers[18])[activeParticleID * 3 + 1] = currK3[1];
						((double *)pointers[18])[activeParticleID * 3 + 2] = currK3[2];
						//k3[activeParticleID * 3] = currK3[0];
						//k3[activeParticleID * 3 + 1] = currK3[1];
						//k3[activeParticleID * 3 + 2] = currK3[2];
					}*/
					((double *)pointers[16])[activeParticleID * 3] = currK[0];
					((double *)pointers[16])[activeParticleID * 3 + 1] = currK[1];
					((double *)pointers[16])[activeParticleID * 3 + 2] = currK[2];

					((double *)pointers[17])[activeParticleID * 3] = currNX[0];
					((double *)pointers[17])[activeParticleID * 3 + 1] = currNX[1];
					((double *)pointers[17])[activeParticleID * 3 + 2] = currNX[2];

					break;
				}

				currCell = nextCell;

				/*double*/ alpha /*exactTime*/ = currTime;
				switch (currStage) {
				case 1:
				case 2: alpha /*exactTime*/ += /*doubleValues[2]*/timeStep * 0.5; break;
				case 3: alpha /*exactTime*/ += /*doubleValues[2]*/timeStep; break;
				}

				/*double*/ alpha = (/*doubleValues[1]*/endTime - alpha/*exactTime*/) / (/*doubleValues[1]*/endTime - /*doubleValues[0]*/startTime);
				/*double*/ beta = 1.0 - alpha;

				/*double vecX[4], vecY[4], vecZ[4];*/

				for (/*int*/ i = 0; i < 4; i++) {
					//int pointID = connectivities[(nextCell << 2) | i];
					nextCell = connectivities[(currCell << 2) | i] * 3;

					vec[i] = startVelocities[nextCell] * alpha + endVelocities[nextCell] * beta;
					vec[i | 4] = startVelocities[nextCell + 1] * alpha + endVelocities[nextCell + 1] * beta;
					vec[i | 8] = startVelocities[nextCell + 2] * alpha + endVelocities[nextCell + 2] * beta;

					//vecX[i] = startVelocities[pointID * 3] * alpha + endVelocities[pointID * 3] * beta;
					//vecY[i] = startVelocities[pointID * 3 + 1] * alpha + endVelocities[pointID * 3 + 1] * beta;
					//vecZ[i] = startVelocities[pointID * 3 + 2] * alpha + endVelocities[pointID * 3 + 2] * beta;
				}

				/*double *currK;*/
				/*
				switch (currStage) {
				case 0: currK = currK1; break;
				case 1: currK = currK2; break;
				case 2: currK = currK3; break;
				case 3: currK = currK4; break;
				}*/

				currK[0] = currK[1] = currK[2] = 0;

				for (/*int*/ i = 0; i < 4; i++) {
					currK[0] += vec[i] * coordinates[i];
					currK[1] += vec[i | 4] * coordinates[i];
					currK[2] += vec[i | 8] * coordinates[i];
				}

				currK[0] *= /*doubleValues[2]*/timeStep;
				currK[1] *= /*doubleValues[2]*/timeStep;
				currK[2] *= /*doubleValues[2]*/timeStep;

				switch (currStage) {
				case 0:
				case 3: alpha = 1.0 / 6; break;
				case 1:
				case 2: alpha = 1.0 / 3; break;
				}

				for (i = 0; i < 3; i++)
					currNX[i] += alpha * currK[i];
				if (currStage == 3) {
					currTime += /*doubleValues[2]*/timeStep;

					for (/*int*/ i = 0; i < 3; i++)
						currLastPosition[i] = currNX[i];
						//currLastPosition[i] += (currK1[i] + 2.0 * (currK2[i] + currK3[i]) + currK4[i]) / 6.0;

					currStage = 0;
				} else
					currStage++;	
			}
		} else break;

	}

}

extern "C"
void InitializeConstantsForBlockedTracingKernelOfRK4(double *globalVertexPositions,
			int *globalTetrahedralConnectivities,
			int *globalTetrahedralLinks,

			int *startOffsetInCell,
			int *startOffsetInPoint,

			double *vertexPositionsForBig,
			double *startVelocitiesForBig,
			double *endVelocitiesForBig,

			int *blockedLocalConnectivities,
			int *blockedLocalLinks,
			int *blockedGlobalCellIDs,

			int *activeBlockList, // Map active block ID to interesting block ID

			int *blockOfGroups,
			int *offsetInBlocks,

			int *stage,
			double *lastPosition,

			double *k, double *nx,

			//double *k2,
			//double *k3,
			double *pastTimes,

			double *placesOfInterest,

			int *startOffsetInParticle,
			int *blockedActiveParticleIDList,
			int *cellLocations,

			int *exitCells,

			double hostTimeStep, double hostEpsilon) {
	int sizeOfPointer = sizeof(void *);

	hipError_t err = (hipError_t)(hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalVertexPositions, sizeOfPointer, 0, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalTetrahedralConnectivities, sizeOfPointer, sizeOfPointer, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalTetrahedralLinks, sizeOfPointer, sizeOfPointer * 2, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInCell, sizeOfPointer, sizeOfPointer * 3, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInPoint, sizeOfPointer, sizeOfPointer * 4, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &vertexPositionsForBig, sizeOfPointer, sizeOfPointer * 5, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startVelocitiesForBig, sizeOfPointer, sizeOfPointer * 6, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &endVelocitiesForBig, sizeOfPointer, sizeOfPointer * 7, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedLocalConnectivities, sizeOfPointer, sizeOfPointer * 8, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedLocalLinks, sizeOfPointer, sizeOfPointer * 9, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedGlobalCellIDs, sizeOfPointer, sizeOfPointer * 10, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &activeBlockList, sizeOfPointer, sizeOfPointer * 11, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockOfGroups, sizeOfPointer, sizeOfPointer * 12, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &offsetInBlocks, sizeOfPointer, sizeOfPointer * 13, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &stage, sizeOfPointer, sizeOfPointer * 14, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &lastPosition, sizeOfPointer, sizeOfPointer * 15, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &k, sizeOfPointer, sizeOfPointer * 16, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &nx, sizeOfPointer, sizeOfPointer * 17, hipMemcpyHostToDevice) |
			  //cudaMemcpyToSymbol(pointers, &k2, sizeOfPointer, sizeOfPointer * 17, cudaMemcpyHostToDevice) |
			  //cudaMemcpyToSymbol(pointers, &k3, sizeOfPointer, sizeOfPointer * 18, cudaMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &pastTimes, sizeOfPointer, sizeOfPointer * 19, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &placesOfInterest, sizeOfPointer, sizeOfPointer * 20, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInParticle, sizeOfPointer, sizeOfPointer * 21, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedActiveParticleIDList, sizeOfPointer, sizeOfPointer * 22, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &cellLocations, sizeOfPointer, sizeOfPointer * 23, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &exitCells, sizeOfPointer, sizeOfPointer * 24, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(timeStep), &hostTimeStep, sizeof(double), 0, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(epsilon), &hostEpsilon, sizeof(double), 0, hipMemcpyHostToDevice));
/*
	err = (cudaError_t)((int)err | cudaMemcpyToSymbol(doubleValues, &startTime, sizeof(double), 0, cudaMemcpyHostToDevice) |
		cudaMemcpyToSymbol(doubleValues, &endTime, sizeof(double), sizeof(double), cudaMemcpyHostToDevice) |
		cudaMemcpyToSymbol(doubleValues, &timeStep, sizeof(double), sizeof(double) * 2, cudaMemcpyHostToDevice) |
		cudaMemcpyToSymbol(doubleValues, &epsilon, sizeof(double), sizeof(double) * 3, cudaMemcpyHostToDevice));
*/
	if (err) {
		printf("Symbol Memcpy Failure\n");
		exit(0);
	}
}

extern "C"
void BlockedTracingOfRK4(/*double *globalVertexPositions,
			int *globalTetrahedralConnectivities,
			int *globalTetrahedralLinks,

			int *startOffsetInCell,
			int *startOffsetInPoint,

			double *vertexPositionsForBig,
			double *startVelocitiesForBig,
			double *endVelocitiesForBig,

			int *blockedLocalConnectivities,
			int *blockedLocalLinks,
			int *blockedGlobalCellIDs,

			int *activeBlockList, // Map active block ID to interesting block ID

			int *blockOfGroups,
			int *offsetInBlocks,

			int *stage,
			double *lastPosition,
			double *k1,
			double *k2,
			double *k3,
			double *pastTimes,

			double *placesOfInterest,

			int *startOffsetInParticle,
			int *blockedActiveParticleIDList,
			int *cellLocations,

			int *exitCells,*/

			double startTime, double endTime, double timeStep, double epsilon, int numOfActiveBlocks,

			int blockSize, int __sharedMemorySize, int __multiple) {
	dim3 dimBlock(blockSize, 1, 1);
	dim3 dimGrid(numOfActiveBlocks, 1, 1);

	hipMemcpyToSymbol(HIP_SYMBOL(sharedMemorySize), &__sharedMemorySize, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(multiple), &__multiple, sizeof(int), 0, hipMemcpyHostToDevice);

#ifdef USE_CACHE
	cudaFuncSetCacheConfig(BlockedTracingKernelOfRK4, cudaFuncCachePreferL1);
#else
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(BlockedTracingKernelOfRK4), hipFuncCachePreferShared);
#endif

#ifdef USE_CACHE
	BlockedTracingKernelOfRK4<<<dimGrid, dimBlock>>>(
#else
	BlockedTracingKernelOfRK4<<<dimGrid, dimBlock, __sharedMemorySize>>>(
#endif
					/*globalVertexPositions,
					globalTetrahedralConnectivities,
					globalTetrahedralLinks,

					startOffsetInCell,
					startOffsetInPoint,

					vertexPositionsForBig,
					startVelocitiesForBig,
					endVelocitiesForBig,

					blockedLocalConnectivities,
					blockedLocalLinks,
					blockedGlobalCellIDs,

					activeBlockList, // Map active block ID to interesting block ID

					blockOfGroups,
					offsetInBlocks,

					stage,
					lastPosition,
					k1,
					k2,
					k3,
					pastTimes,

					placesOfInterest,

					startOffsetInParticle,
					blockedActiveParticleIDList,
					cellLocations,

					exitCells,
*/
					startTime, endTime, timeStep, epsilon //, sharedMemorySize, multiple
					);

	hipError_t err = hipDeviceSynchronize();
	if (err) {
		printf("err = %d\n", err);
		hipGetErrorString(err);
		exit(0);
	}
}
