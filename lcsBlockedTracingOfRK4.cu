/*****************************************************
File		:	lcsBlockedTracingOfRK4.cu
Author		:	Mingcheng Chen
Last Update	:	February 14th, 2013
******************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define MINGCHENG_CHEN


__device__ inline double DeterminantThree(double *a) {
	// a[0] a[1] a[2]
	// a[3] a[4] a[5]
	// a[6] a[7] a[8]
	//return a[0] * a[4] * a[8] + a[1] * a[5] * a[6] + a[2] * a[3] * a[7] -
	//       a[0] * a[5] * a[7] - a[1] * a[3] * a[8] - a[2] * a[4] * a[6];
	return a[0] * (a[4] * a[8] - a[5] * a[7]) + a[1] * (a[5] * a[6] - a[3] * a[8]) + a[2] * (a[3] * a[7] - a[4] * a[6]);
}

#ifndef MINGCHENG_CHEN
__device__ void CalculateNaturalCoordinates(double X, double Y, double Z, double *tetX, double *tetY, double *tetZ, double *coordinates) {
	double x0 = tetX[0];
	double y0 = tetY[0];
	double z0 = tetZ[0];

	double x1 = tetX[1];
	double y1 = tetY[1];
	double z1 = tetZ[1];

	double x2 = tetX[2];
	double y2 = tetY[2];
	double z2 = tetZ[2];

	double x3 = tetX[3];
	double y3 = tetY[3];
	double z3 = tetZ[3];

	// Determinant of mapping from natural to physical coordinates of test element
	double V = (x1 - x0) * ((y2 - y0) * (z3 - z0) - (z2 - z0) * (y3 - y0)) +
		(x2 - x0) * ((y0 - y1) * (z3 - z0) - (z0 - z1) * (y3 - y0)) +
		(x3 - x0) * ((y1 - y0) * (z2 - z0) - (z1 - z0) * (y2 - y0));	

	// Natural coordinates of point to be interpolated
	coordinates[1] = ((((z3 - z0) * (y2 - y3) - (z2 - z3) * (y3 - y0)) * (X - x0)) + 
			  (((x3 - x0) * (z2 - z3) - (x2 - x3) * (z3 - z0)) * (Y - y0)) +
			  (((y3 - y0) * (x2 - x3) - (y2 - y3) * (x3 - x0)) * (Z - z0))
                         ) / V;
			
	coordinates[2] = ((((z3 - z0) * (y0 - y1) - (z0 - z1) * (y3 - y0)) * (X - x0)) +
			  (((x3 - x0) * (z0 - z1) - (x0 - x1) * (z3 - z0)) * (Y - y0)) +
			  (((y3 - y0) * (x0 - x1) - (y0 - y1) * (x3 - x0)) * (Z - z0))
			 ) / V;
			
	coordinates[3] = ((((z1 - z2) * (y0 - y1) - (z0 - z1) * (y1 - y2)) * (X - x0)) +
			  (((x1 - x2) * (z0 - z1) - (x0 - x1) * (z1 - z2)) * (Y - y0)) +
			  (((y1 - y2) * (x0 - x1) - (y0 - y1) * (x1 - x2)) * (Z - z0))
			 ) / V;

	coordinates[0] = 1.0 - coordinates[1] - coordinates[2] - coordinates[3];		
}
#endif

#ifdef MINGCHENG_CHEN
__device__ inline void CalculateNaturalCoordinates(double X, double Y, double Z,
					double *tetX, double *tetY, double *tetZ, double *coordinates) {
	X -= tetX[0];
	Y -= tetY[0];
	Z -= tetZ[0];

	double det[9] = {tetX[1] - tetX[0], tetY[1] - tetY[0], tetZ[1] - tetZ[0],
			 tetX[2] - tetX[0], tetY[2] - tetY[0], tetZ[2] - tetZ[0],
			 tetX[3] - tetX[0], tetY[3] - tetY[0], tetZ[3] - tetZ[0]};

	double V = 1.0 / DeterminantThree(det);

	double z41 = tetZ[3] - tetZ[0];
	double y34 = tetY[2] - tetY[3];
	double z34 = tetZ[2] - tetZ[3];
	double y41 = tetY[3] - tetY[0];
	double a11 = z41 * y34 - z34 * y41;

	double x41 = tetX[3] - tetX[0];
	double x34 = tetX[2] - tetX[3];
	double a12 = x41 * z34 - x34 * z41;

	double a13 = y41 * x34 - y34 * x41;

	coordinates[1] = (a11 * X + a12 * Y + a13 * Z) * V;

	double y12 = tetY[0] - tetY[1];
	double z12 = tetZ[0] - tetZ[1];
	double a21 = z41 * y12 - z12 * y41;

	double x12 = tetX[0] - tetX[1];
	double a22 = x41 * z12 - x12 * z41;

	double a23 = y41 * x12 - y12 * x41;

	coordinates[2] = (a21 * X + a22 * Y + a23 * Z) * V;

	double z23 = tetZ[1] - tetZ[2];
	double y23 = tetY[1] - tetY[2];
	double a31 = z23 * y12 - z12 * y23;

	double x23 = tetX[1] - tetX[2];
	double a32 = x23 * z12 - x12 * z23;

	double a33 = y23 * x12 - y12 * x23;

	coordinates[3] = (a31 * X + a32 * Y + a33 * Z) * V;

	coordinates[0] = 1.0 - coordinates[1] - coordinates[2] - coordinates[3];
}
#endif

__device__ inline int FindCell(double *particle, int *connectivities, int *links, double *vertexPositions,
			double epsilon, int guess, double *coordinates) {
	double tetX[4], tetY[4], tetZ[4];

	while (true) {
		for (int i = 0; i < 4; i++) {
			int pointID = connectivities[(guess << 2) | i];

			tetX[i] = vertexPositions[pointID * 3];
			tetY[i] = vertexPositions[pointID * 3 + 1];
			tetZ[i] = vertexPositions[pointID * 3 + 2];
		}

		CalculateNaturalCoordinates(particle[0], particle[1], particle[2], tetX, tetY, tetZ, coordinates);
		
		int index = 0;

		for (int i = 1; i < 4; i++)
			if (coordinates[i] < coordinates[index]) index = i;
		if (coordinates[index] >= -epsilon) break;

		guess = links[(guess << 2) | index];
		
		if (guess == -1) break;
	}

	return guess;
}

__constant__ void *pointers[25];

__global__ void BlockedTracingKernelOfRK4(/*double *globalVertexPositions,
					int *globalTetrahedralConnectivities,
					int *globalTetrahedralLinks,

					int *startOffsetInCell,
					int *startOffsetInPoint,

					double *vertexPositionsForBig,
					double *startVelocitiesForBig,
					double *endVelocitiesForBig,

					int *blockedLocalConnectivities,
					int *blockedLocalLinks,
					int *blockedGlobalCellIDs,

					int *activeBlockList, // Map active block ID to interesting block ID

					int *blockOfGroups,
					int *offsetInBlocks,

					int *stage,
					double *lastPosition,
					double *k1,
					double *k2,
					double *k3,
					double *pastTimes,

					double *placesOfInterest,

					int *startOffsetInParticle,
					int *blockedActiveParticleIDList,
					int *cellLocations,

					int *exitCells,
*/
					double startTime, double endTime, double timeStep, double epsilon,

					int sharedMemorySize, int multiple) {
/*
	cudaError_t err = cudaMemcpyToSymbol(pointers, &globalVertexPositions, sizeOfPointer, 0, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &globalTetrahedralConnectivities, sizeOfPointer, sizeOfPointer, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &globalTetrahedralLinks, sizeOfPointer, sizeOfPointer * 2, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInCell, sizeOfPointer, sizeOfPointer * 3, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInPoint, sizeOfPointer, sizeOfPointer * 4, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &vertexPositionsForBig, sizeOfPointer, sizeOfPointer * 5, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startVelocitiesForBig, sizeOfPointer, sizeOfPointer * 6, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &endVelocitiesForBig, sizeOfPointer, sizeOfPointer * 7, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedLocalConnectivities, sizeOfPointer, sizeOfPointer * 8, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedLocalLinks, sizeOfPointer, sizeOfPointer * 9, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedGlobalCellIDs, sizeOfPointer, sizeOfPointer * 10, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &activeBlockList, sizeOfPointer, sizeOfPointer * 11, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockOfGroups, sizeOfPointer, sizeOfPointer * 12, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &offsetInBlocks, sizeOfPointer, sizeOfPointer * 13, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &stage, sizeOfPointer, sizeOfPointer * 14, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &lastPosition, sizeOfPointer, sizeOfPointer * 15, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k1, sizeOfPointer, sizeOfPointer * 16, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k2, sizeOfPointer, sizeOfPointer * 17, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &k3, sizeOfPointer, sizeOfPointer * 18, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &pastTimes, sizeOfPointer * 19, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &placesOfInterest, sizeOfPointer, sizeOfPointer * 20, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &startOffsetInParticle, sizeOfPointer, sizeOfPointer * 21, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &blockedActiveParticleIDList, sizeOfPointer, sizeOfPointer * 22, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &cellLocations, sizeOfPointer, sizeOfPointer * 23, cudaMemcpyHostToDevice) |
			  cudaMemcpyToSymbol(pointers, &exitCells, sizeOfPointer, sizeOfPointer * 24, cudaMemcpyHostToDevice) |
*/


	__shared__ extern char sharedMemory[];
	//char *sharedMemory;

	// Get work group ID
	//int groupID = blockIdx.x;
	
	// Get number of threads in a work group
	//int numOfThreads = blockDim.x;

	// Get local thread ID
	//int localID = threadIdx.x;

	// Get active block ID
	int activeBlockID = ((int *)pointers[12])[blockIdx.x/*groupID*/];
	//int activeBlockID = blockOfGroups[groupID];

	// Get interesting block ID of the work group
	int interestingBlockID = ((int *)pointers[11])[activeBlockID];
	//int interestingBlockID = activeBlockList[activeBlockID];

	// Declare some arrays
	double *vertexPositions;
	double *startVelocities;
	double *endVelocities;
	int *connectivities;
	int *links;

	int startCell = ((int *)pointers[3])[interestingBlockID];
	int startPoint = ((int *)pointers[4])[interestingBlockID];
	//int startCell = startOffsetInCell[interestingBlockID];
	//int startPoint = startOffsetInPoint[interestingBlockID];

	int numOfCells = ((int *)pointers[3])[interestingBlockID + 1] - startCell;
	int numOfPoints = ((int *)pointers[4])[interestingBlockID + 1] - startPoint;
	//int numOfCells = startOffsetInCell[interestingBlockID + 1] - startCell;
	//int numOfPoints = startOffsetInPoint[interestingBlockID + 1] - startPoint;

	// Assuming int is 4 bytes and double is 8 bytes
	//	 localNumOfCells * sizeof(int) * 4 +		// this->localConnectivities
	//	 localNumOfCells * sizeof(int) * 4 +		// this->localLinks
	//	 localNumOfPoints * sizeof(double) * 3 +	// point positions
	//	 localNumOfPoints * sizeof(double) * 3 * 2;	// point velocities (start and end)

	if (((numOfCells << 5) + ((numOfPoints * 9) << 3)) <= sharedMemorySize) { // This branch fills in the shared memory
		// Initialize vertexPositions, startVelocities and endVelocities
		vertexPositions = (double *)sharedMemory;
		startVelocities = vertexPositions + numOfPoints * 3;
		endVelocities = startVelocities + numOfPoints * 3;

		// Initialize connectivities and links
		connectivities = (int *)(endVelocities + numOfPoints * 3);
		links = connectivities + (numOfCells << 2);

		for (int i = threadIdx.x/*localID*/; i < numOfPoints * 3; i += blockDim.x/*numOfThreads*/) {
			vertexPositions[i] = ((double *)pointers[5])[startPoint * 3 + i];
			startVelocities[i] = ((double *)pointers[6])[startPoint * 3 + i];
			endVelocities[i] = ((double *)pointers[7])[startPoint * 3 + i];
			//vertexPositions[i] = vertexPositionsForBig[startPoint * 3 + i];
			//startVelocities[i] = startVelocitiesForBig[startPoint * 3 + i];
			//endVelocities[i] = endVelocitiesForBig[startPoint * 3 + i];
		}

		for (int i = threadIdx.x/*localID*/; i < (numOfCells << 2); i += blockDim.x/*numOfThreads*/) {
			connectivities[i] = ((int *)pointers[8])[(startCell << 2) + i];
			links[i] = ((int *)pointers[9])[(startCell << 2) + i];
			//connectivities[i] = blockedLocalConnectivities[(startCell << 2) + i];
			//links[i] = blockedLocalLinks[(startCell << 2) + i];
		}

		//__syncthreads();
	} else { // This branch fills in the global memory
		// Initialize vertexPositions, startVelocities and endVelocities
		vertexPositions = (double *)pointers[5] + startPoint * 3;
		startVelocities = (double *)pointers[6] + startPoint * 3;
		endVelocities = (double *)pointers[7] + startPoint * 3;
		//vertexPositions = vertexPositionsForBig + startPoint * 3;
		//startVelocities = startVelocitiesForBig + startPoint * 3;
		//endVelocities = endVelocitiesForBig + startPoint * 3;

		// Initialize connectivities and links
		connectivities = (int *)pointers[8] + (startCell << 2);
		links = (int *)pointers[9] + (startCell << 2);
		//connectivities = blockedLocalConnectivities + (startCell << 2);
		//links = blockedLocalLinks + (startCell << 2);
	}

	__syncthreads();

	int numOfActiveParticles = ((int *)pointers[21])[activeBlockID + 1] - ((int *)pointers[21])[activeBlockID];
	int offset = ((int *)pointers[13])[blockIdx.x/*groupID*/] * blockDim.x/*numOfThreads*/ * multiple;
	//int numOfActiveParticles = startOffsetInParticle[activeBlockID + 1] - startOffsetInParticle[activeBlockID];
	//int offset = offsetInBlocks[groupID] * numOfThreads * multiple;

	int idx, activeParticleID, currStage, currCell, nextCell;
	double currTime;
	double currLastPosition[3], currK1[3], currK2[3], currK3[3], currK4[3];
	double placeOfInterest[3];
	double coordinates[4];

	for (idx = threadIdx.x/*localID*/; idx < blockDim.x/*numOfThreads*/ * multiple; idx += blockDim.x/*numOfThreads*/) {
		//int arrayIdx = offsetInBlocks[groupID] * numOfThreads + localID;
		activeParticleID = offset + idx;
		//int arrayIdx = offset + idx;

		//if (arrayIdx < numOfActiveParticles) {
		if (activeParticleID < numOfActiveParticles) {
			// activeParticleID here means the initial active particle ID
			//arrayIdx += ((int *)pointers[21])[activeBlockID];
			//int activeParticleID = ((int *)pointers[22])[arrayIdx];
			activeParticleID = ((int *)pointers[22])[activeParticleID + ((int *)pointers[21])[activeBlockID]];
			//arrayIdx += startOffsetInParticle[activeBlockID];
			//int activeParticleID = blockedActiveParticleIDList[arrayIdx];

			// Initialize the particle status
			/*int*/ currStage = ((int *)pointers[14])[activeParticleID];
			/*int*/ currCell = ((int *)pointers[23])[activeParticleID];
			//int currStage = stage[activeParticleID];
			//int currCell = cellLocations[activeParticleID];

			/*double*/ currTime = ((double *)pointers[19])[activeParticleID];
			//double currTime = pastTimes[activeParticleID];

			/*double currLastPosition[3];*/
			currLastPosition[0] = ((double *)pointers[15])[activeParticleID * 3];
			currLastPosition[1] = ((double *)pointers[15])[activeParticleID * 3 + 1];
			currLastPosition[2] = ((double *)pointers[15])[activeParticleID * 3 + 2];
			//currLastPosition[0] = lastPosition[activeParticleID * 3];
			//currLastPosition[1] = lastPosition[activeParticleID * 3 + 1];
			//currLastPosition[2] = lastPosition[activeParticleID * 3 + 2];
			/*double currK1[3], currK2[3], currK3[3], currK4[3];*/
			if (currStage > 0) {
				currK1[0] = ((double *)pointers[16])[activeParticleID * 3];
				currK1[1] = ((double *)pointers[16])[activeParticleID * 3 + 1];
				currK1[2] = ((double *)pointers[16])[activeParticleID * 3 + 2];
				//currK1[0] = k1[activeParticleID * 3];
				//currK1[1] = k1[activeParticleID * 3 + 1];
				//currK1[2] = k1[activeParticleID * 3 + 2];
			}
			if (currStage > 1) {
				currK2[0] = ((double *)pointers[17])[activeParticleID * 3];
				currK2[1] = ((double *)pointers[17])[activeParticleID * 3 + 1];
				currK2[2] = ((double *)pointers[17])[activeParticleID * 3 + 2];
				//currK2[0] = k2[activeParticleID * 3];
				//currK2[1] = k2[activeParticleID * 3 + 1];
				//currK2[2] = k2[activeParticleID * 3 + 2];
			}
			if (currStage > 2) {
				currK3[0] = ((double *)pointers[18])[activeParticleID * 3];
				currK3[1] = ((double *)pointers[18])[activeParticleID * 3 + 1];
				currK3[2] = ((double *)pointers[18])[activeParticleID * 3 + 2];
				//currK3[0] = k3[activeParticleID * 3];
				//currK3[1] = k3[activeParticleID * 3 + 1];
				//currK3[2] = k3[activeParticleID * 3 + 2];
			}

			// At least one loop is executed.
			while (true) {
				/*double placeOfInterest[3];*/
				placeOfInterest[0] = currLastPosition[0];
				placeOfInterest[1] = currLastPosition[1];
				placeOfInterest[2] = currLastPosition[2];
				switch (currStage) {
				case 1: {
					placeOfInterest[0] += 0.5 * currK1[0];
					placeOfInterest[1] += 0.5 * currK1[1];
					placeOfInterest[2] += 0.5 * currK1[2];
					} break;
				case 2: {
					placeOfInterest[0] += 0.5 * currK2[0];
					placeOfInterest[1] += 0.5 * currK2[1];
					placeOfInterest[2] += 0.5 * currK2[2];
					} break;
				case 3: {
					placeOfInterest[0] += currK3[0];
					placeOfInterest[1] += currK3[1];
					placeOfInterest[2] += currK3[2];
					} break;
				}

				/*double coordinates[4];*/

				/*int*/ nextCell = FindCell(placeOfInterest, connectivities, links, vertexPositions, epsilon, currCell, coordinates);

				if (nextCell == -1 || currTime >= endTime) {
					// Find the next cell globally
					int globalCellID = ((int *)pointers[10])[startCell + currCell];
					//int globalCellID = blockedGlobalCellIDs[startCell + currCell];
					int nextGlobalCell;
				
					if (nextCell != -1)
						nextGlobalCell = ((int *)pointers[10])[startCell + nextCell];
						//nextGlobalCell = blockedGlobalCellIDs[startCell + nextCell];
					else
						nextGlobalCell = FindCell(placeOfInterest, (int *)pointers[1], (int *)pointers[2], (double *)pointers[0], epsilon, globalCellID, coordinates);
						//nextGlobalCell = FindCell(placeOfInterest, globalTetrahedralConnectivities,
						//			globalTetrahedralLinks, globalVertexPositions,
						//			epsilon, globalCellID, coordinates);

					if (currTime >= endTime && nextGlobalCell != -1) nextGlobalCell = -2 - nextGlobalCell;

					((double *)pointers[19])[activeParticleID] = currTime;
					//pastTimes[activeParticleID] = currTime;

					((int *)pointers[14])[activeParticleID] = currStage;
					//stage[activeParticleID] = currStage;

					((double *)pointers[15])[activeParticleID * 3] = currLastPosition[0];
					((double *)pointers[15])[activeParticleID * 3 + 1] = currLastPosition[1];
					((double *)pointers[15])[activeParticleID * 3 + 2] = currLastPosition[2];
					//lastPosition[activeParticleID * 3] = currLastPosition[0];
					//lastPosition[activeParticleID * 3 + 1] = currLastPosition[1];
					//lastPosition[activeParticleID * 3 + 2] = currLastPosition[2];

					((double *)pointers[20])[activeParticleID * 3] = placeOfInterest[0];
					((double *)pointers[20])[activeParticleID * 3 + 1] = placeOfInterest[1];
					((double *)pointers[20])[activeParticleID * 3 + 2] = placeOfInterest[2];
					//placesOfInterest[activeParticleID * 3] = placeOfInterest[0];
					//placesOfInterest[activeParticleID * 3 + 1] = placeOfInterest[1];
					//placesOfInterest[activeParticleID * 3 + 2] = placeOfInterest[2];

					((int *)pointers[24])[activeParticleID] = nextGlobalCell;
					//exitCells[activeParticleID] = nextGlobalCell;
		
					if (currStage > 0) {
						((double *)pointers[16])[activeParticleID * 3] = currK1[0];
						((double *)pointers[16])[activeParticleID * 3 + 1] = currK1[1];
						((double *)pointers[16])[activeParticleID * 3 + 2] = currK1[2];
						//k1[activeParticleID * 3] = currK1[0];
						//k1[activeParticleID * 3 + 1] = currK1[1];
						//k1[activeParticleID * 3 + 2] = currK1[2];
					}
					if (currStage > 1) {
						((double *)pointers[17])[activeParticleID * 3] = currK2[0];
						((double *)pointers[17])[activeParticleID * 3 + 1] = currK2[1];
						((double *)pointers[17])[activeParticleID * 3 + 2] = currK2[2];
						//k2[activeParticleID * 3] = currK2[0];
						//k2[activeParticleID * 3 + 1] = currK2[1];
						//k2[activeParticleID * 3 + 2] = currK2[2];
					}
					if (currStage > 2) {
						((double *)pointers[18])[activeParticleID * 3] = currK3[0];
						((double *)pointers[18])[activeParticleID * 3 + 1] = currK3[1];
						((double *)pointers[18])[activeParticleID * 3 + 2] = currK3[2];
						//k3[activeParticleID * 3] = currK3[0];
						//k3[activeParticleID * 3 + 1] = currK3[1];
						//k3[activeParticleID * 3 + 2] = currK3[2];
					}
					break;
				}

				currCell = nextCell;

				double exactTime = currTime;
				switch (currStage) {
				case 0: break;
				case 1:
				case 2: exactTime += timeStep * 0.5; break;
				case 3: exactTime += timeStep; break;
				}

				double alpha = (endTime - exactTime) / (endTime - startTime);
				double beta = 1 - alpha;

				double vecX[4], vecY[4], vecZ[4];

				for (int i = 0; i < 4; i++) {
					int pointID = connectivities[(nextCell << 2) | i];
					vecX[i] = startVelocities[pointID * 3] * alpha + endVelocities[pointID * 3] * beta;
					vecY[i] = startVelocities[pointID * 3 + 1] * alpha + endVelocities[pointID * 3 + 1] * beta;
					vecZ[i] = startVelocities[pointID * 3 + 2] * alpha + endVelocities[pointID * 3 + 2] * beta;
				}

				double *currK;
				switch (currStage) {
				case 0: currK = currK1; break;
				case 1: currK = currK2; break;
				case 2: currK = currK3; break;
				case 3: currK = currK4; break;
				}

				currK[0] = currK[1] = currK[2] = 0;

				for (int i = 0; i < 4; i++) {
					currK[0] += vecX[i] * coordinates[i];
					currK[1] += vecY[i] * coordinates[i];
					currK[2] += vecZ[i] * coordinates[i];
				}

				currK[0] *= timeStep;
				currK[1] *= timeStep;
				currK[2] *= timeStep;

				if (currStage == 3) {
					currTime += timeStep;

					for (int i = 0; i < 3; i++)
						currLastPosition[i] += (currK1[i] + 2 * currK2[i] + 2 * currK3[i] + currK4[i]) / 6;

					currStage = 0;
				} else
					currStage++;
			}
		} else break;

	}

}

extern "C"
void BlockedTracingOfRK4(double *globalVertexPositions,
			int *globalTetrahedralConnectivities,
			int *globalTetrahedralLinks,

			int *startOffsetInCell,
			int *startOffsetInPoint,

			double *vertexPositionsForBig,
			double *startVelocitiesForBig,
			double *endVelocitiesForBig,

			int *blockedLocalConnectivities,
			int *blockedLocalLinks,
			int *blockedGlobalCellIDs,

			int *activeBlockList, // Map active block ID to interesting block ID

			int *blockOfGroups,
			int *offsetInBlocks,

			int *stage,
			double *lastPosition,
			double *k1,
			double *k2,
			double *k3,
			double *pastTimes,

			double *placesOfInterest,

			int *startOffsetInParticle,
			int *blockedActiveParticleIDList,
			int *cellLocations,

			int *exitCells,

			double startTime, double endTime, double timeStep, double epsilon, int numOfActiveBlocks,

			int blockSize, int sharedMemorySize, int multiple) {
	dim3 dimBlock(blockSize, 1, 1);
	dim3 dimGrid(numOfActiveBlocks, 1, 1);

	int sizeOfPointer = sizeof(void *);

	/*
	/// DEBUG ///
	printf("sizeOfPointer = %d\n", sizeOfPointer);
	printf("sizeof(long long) = %d\n", sizeof(long long));

	printf("globalVertexPositions = %lld\n", (long long)globalVertexPositions);
	printf("activeBlockList = %lld\n", (long long)activeBlockList);
	printf("pastTimes = %lld\n", (long long)pastTimes);
	*/

	hipError_t err = (hipError_t)(hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalVertexPositions, sizeOfPointer, 0, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalTetrahedralConnectivities, sizeOfPointer, sizeOfPointer, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &globalTetrahedralLinks, sizeOfPointer, sizeOfPointer * 2, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInCell, sizeOfPointer, sizeOfPointer * 3, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInPoint, sizeOfPointer, sizeOfPointer * 4, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &vertexPositionsForBig, sizeOfPointer, sizeOfPointer * 5, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startVelocitiesForBig, sizeOfPointer, sizeOfPointer * 6, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &endVelocitiesForBig, sizeOfPointer, sizeOfPointer * 7, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedLocalConnectivities, sizeOfPointer, sizeOfPointer * 8, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedLocalLinks, sizeOfPointer, sizeOfPointer * 9, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedGlobalCellIDs, sizeOfPointer, sizeOfPointer * 10, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &activeBlockList, sizeOfPointer, sizeOfPointer * 11, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockOfGroups, sizeOfPointer, sizeOfPointer * 12, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &offsetInBlocks, sizeOfPointer, sizeOfPointer * 13, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &stage, sizeOfPointer, sizeOfPointer * 14, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &lastPosition, sizeOfPointer, sizeOfPointer * 15, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &k1, sizeOfPointer, sizeOfPointer * 16, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &k2, sizeOfPointer, sizeOfPointer * 17, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &k3, sizeOfPointer, sizeOfPointer * 18, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &pastTimes, sizeOfPointer, sizeOfPointer * 19, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &placesOfInterest, sizeOfPointer, sizeOfPointer * 20, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &startOffsetInParticle, sizeOfPointer, sizeOfPointer * 21, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &blockedActiveParticleIDList, sizeOfPointer, sizeOfPointer * 22, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &cellLocations, sizeOfPointer, sizeOfPointer * 23, hipMemcpyHostToDevice) |
			  hipMemcpyToSymbol(HIP_SYMBOL(pointers), &exitCells, sizeOfPointer, sizeOfPointer * 24, hipMemcpyHostToDevice));
	if (err) {
		hipGetErrorString(err);
		printf("Symbol\n");
		exit(0);
	}

	BlockedTracingKernelOfRK4<<<dimGrid, dimBlock, sharedMemorySize>>>(/*globalVertexPositions,
					globalTetrahedralConnectivities,
					globalTetrahedralLinks,

					startOffsetInCell,
					startOffsetInPoint,

					vertexPositionsForBig,
					startVelocitiesForBig,
					endVelocitiesForBig,

					blockedLocalConnectivities,
					blockedLocalLinks,
					blockedGlobalCellIDs,

					activeBlockList, // Map active block ID to interesting block ID

					blockOfGroups,
					offsetInBlocks,

					stage,
					lastPosition,
					k1,
					k2,
					k3,
					pastTimes,

					placesOfInterest,

					startOffsetInParticle,
					blockedActiveParticleIDList,
					cellLocations,

					exitCells,
*/
					startTime, endTime, timeStep, epsilon,

					sharedMemorySize, multiple);

	err = hipDeviceSynchronize();
	if (err) {
		printf("err = %d\n", err);
		hipGetErrorString(err);
		exit(0);
	}
}
